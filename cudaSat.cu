#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <limits.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define THREADS 512

/* Polygon 'Struct' */
static float * polygon_x;
static float * polygon_y;
static float ** polygon_vertices; // fixed to size [polygons][8]
static int * polygon_num_vertices;

/* Contact 'Struct' */
static int * contact_p1;
static int * contact_p2;
static float * contact_n_x; /* normal axis to the polygon */
static float * contact_n_y;
static float * contact_penetration;
static int * contact_used_flag;

static int num_polygons, num_threads, num_contacts, num_vertices;
void pthread_init(float * p_x, float * p_y, int * c_p1, int * c_p2, float * c_n_x, float * c_n_y, float * c_p, int * c_u_f, int n_polygons, int n_contacts, int n_threads);

void createPolygons() {
    int i = 0;
    for(i = 0; i < num_polygons; i++) {
        float x = 5; // generate position- needs to be random
        float y = 5; // generate position- needs to be random
        polygon_x[i] = i * 10 + x;
        polygon_y[i] = i * 10 + y;
        polygon_num_vertices[i] = num_vertices;
        polygon_vertices[i] = (float *) malloc(sizeof(float) * num_vertices * 2);

        /* make a square */
        int size = rand() % 10 + 3;
        polygon_vertices[i][0] = polygon_x[i];
        polygon_vertices[i][1] = polygon_y[i];

        polygon_vertices[i][2] = polygon_x[i]+size;
        polygon_vertices[i][3] = polygon_y[i];

        polygon_vertices[i][4] = polygon_x[i]+size;
        polygon_vertices[i][5] = polygon_y[i]-size;

        polygon_vertices[i][6] = polygon_x[i];
        polygon_vertices[i][7] = polygon_y[i]-size;
    }
}

void printPolygons() {
    int i;
    for(i = 0; i < num_polygons; i++) {
        printf("Polygon %d: %f %f %d\n", i, polygon_x[i], polygon_y[i], polygon_num_vertices[i]);
    }
}

__global__ void detectCollisions(int num_polygons, float * polygon_x, float * polygon_y, 
                                 int * polygon_num_vertices, float ** polygon_vertices, int * contact_p1, int * contact_p2,
                                 float * contact_n_x, float * contact_n_y, float * contact_penetration, int * contact_used_flag) {
    register int i, j;
    int rank = threadIdx.x + blockIdx.x * blockDim.x;


    if(rank < num_polygons) {
        for(j = i+1; j < num_polygons; j++) { // prevents duplicate checks- each polygon only checks the one behind it in the list.

            // get edges
            float i_edges[4 * 2];

            int k;
            for(k = 0; k < 4 * 2 - 2; k+=2) {
                float v2x = polygon_vertices[i][k+2];
                float v2y = polygon_vertices[i][k+3];
                float v1x = polygon_vertices[i][k];
                float v1y = polygon_vertices[i][k+1];

                float e_x = v2x - v1x;
                float e_y = v2y - v1y;

                i_edges[k] = e_x;
                i_edges[k+1] = e_y;
            }

            // add the last edge- last vertice to the first
            float v2x = polygon_vertices[i][k];
            float v2y = polygon_vertices[i][k+1];
            float v1x = polygon_vertices[i][0];
            float v1y = polygon_vertices[i][1];

            float e_x = v2x - v1x;
            float e_y = v2y - v1y;

            i_edges[k] = e_x;
            i_edges[k+1] = e_y;            

            float j_edges[4 * 2];

            for(k = 0; k < 4*2-2; k+=2) {
                float v2x = polygon_vertices[j][k+2];
                float v2y = polygon_vertices[j][k+3];
                float v1x = polygon_vertices[j][k];
                float v1y = polygon_vertices[j][k+1];

                float e_x = v2x - v1x;
                float e_y = v2y - v1y;

                j_edges[k] = e_x;
                j_edges[k+1] = e_y;
            }

            // add the last edge- last vertice to the first
            v2x = polygon_vertices[j][k];
            v2y = polygon_vertices[j][k+1];
            v1x = polygon_vertices[j][0];
            v1y = polygon_vertices[j][1];

            e_x = v2x - v1x;
            e_y = v2y - v1y;

            j_edges[k] = e_x;
            j_edges[k+1] = e_y;

            // printf("2!\n");

            // merge i_edges and j_edges
            float num_edges = (4*2 + 4*2);
            float edges[4*2 + 4*2];
            memcpy(edges, i_edges, sizeof(float) * 4*2);
            memcpy(edges + (4*2), j_edges, sizeof(float) * 4*2);

            float min_overlap = FLT_MAX;
            float min_axis[2];
            int collision = 1; // True
            for(k = 0; k < num_edges; k+=2) {
                float i_proj[2], j_proj[2];

                // perp vector
                float axis[2];
                axis[0] = -1 * edges[k+1];
                axis[1] = edges[k];

                // normalize vector
                float esp = .0000001; // prevent division by zero by adding trivial amount
                float lengthsq = axis[0] * axis[0] + axis[1] * axis[1];
                /*
                axis[0] = axis[0] * rsqrtf(lengthsq + esp);
                axis[1] = axis[1] * rsqrtf(lengthsq + esp);
                */

                axis[0] = axis[0] / sqrtf(lengthsq + esp);
                axis[1] = axis[1] / sqrtf(lengthsq + esp);

                // project each polygon onto the axis
                float max = -FLT_MAX;
                float min = FLT_MAX;
                int l;
                for(l = 0; l < 4*2; l+=2) {
                    int res = polygon_vertices[i][l] * axis[0] + polygon_vertices[i][l+1] * axis[1];
                    if(max < res) { max = res; }
                    if(min > res) { min = res; }
                }

                i_proj[0] = min;
                i_proj[1] = max;

                max = -FLT_MAX;
                min = FLT_MAX;
                for(l = 0; l < 4*2; l+=2) {
                    int res = polygon_vertices[j][l] * axis[0] + polygon_vertices[j][l+1] * axis[1];
                    if(max < res) { max = res; }
                    if(min > res) { min = res; }
                }

                j_proj[0] = min;
                j_proj[1] = max;

                // check for overlap- determines the overlap of two line segments
                float overlap;
                if(i_proj[0] < j_proj[0])
                    overlap = j_proj[0] - i_proj[1];
                else
                    overlap = i_proj[0] - j_proj[1];

                if(overlap > 0) {
                    collision = 0;
                    break;
                }

                if(overlap < min_overlap) {
                    min_overlap = overlap;
                    min_axis[0] = axis[0];
                    min_axis[1] = axis[1];
                }
            }

            if(collision == 1) {
                int index = rank * num_polygons + j;
                contact_p1[index] = i;
                contact_p2[index] = j;
                contact_n_x[index] = min_axis[0];
                contact_n_y[index] = min_axis[1];
                contact_penetration[index] = min_overlap;
                contact_used_flag[index] = 1;
            }

            free(i_edges);
            free(j_edges);
            free(edges);
        }
    }
}


int main(int argc, char * argv[]) {
    register int blocks;
    /* Initialize */
    srand(time(NULL));
    register int i;
    struct timeval start, end;
    /* check command line */
    if (argc != 3) {
        fprintf(stderr, "usage: %s number_of_polygons number_of_threads\n", argv[0]);
        exit(-1);
    }

    num_polygons = atoi(argv[1]);
    num_threads = atoi(argv[2]);
    num_contacts = num_polygons * num_polygons;
    num_vertices = 4;

    printf("Separating Axis v1.0: %d polygons %d threads %d \n", num_polygons, num_threads, num_vertices);

    /* Allocate Arrays */
    polygon_x = (float *) malloc(sizeof(float) * num_polygons); 
    polygon_y = (float *) malloc(sizeof(float) * num_polygons);
    polygon_num_vertices = (int *) malloc(sizeof(int) * num_polygons);
    polygon_vertices = (float **) malloc(sizeof(float*) * num_polygons);
    

    contact_p1 = (int *) malloc(sizeof(int) * num_contacts);
    contact_p2 = (int *) malloc(sizeof(int) * num_contacts);
    contact_n_x = (float *) malloc(sizeof(float) * num_contacts);
    contact_n_y = (float *) malloc(sizeof(float) * num_contacts);
    contact_penetration = (float *) malloc(sizeof(float) * num_contacts);
    contact_used_flag = (int *) malloc(sizeof(int) * num_contacts);

    for(i = 0; i < num_contacts; i++) { contact_used_flag[i] = 0; }    


    /* Generate Polygons */
    createPolygons();

    // printPolygons();

    /* Start Time */
    gettimeofday(&start, NULL);

    // /* CUDA kernel invocaiton */
    blocks = (num_polygons + THREADS - 1) / THREADS;
    float *pd_x, *pd_y, **pdvert;
    int *pd_num_vert;
    int *cd_p1; int *cd_p2; float *cd_n_x; float *cd_n_y; float *cd_pen; int *cd_used_flag;

    if (hipSuccess != hipMalloc((void **)&pd_x, num_polygons * sizeof(float))) fprintf(stderr, "could not allocate array\n");
    if (hipSuccess != hipMalloc((void **)&pd_y, num_polygons * sizeof(float))) fprintf(stderr, "could not allocate array\n");
    if (hipSuccess != hipMalloc((void **)&pd_num_vert, num_polygons * sizeof(float))) fprintf(stderr, "could not allocate array\n");
    if (hipSuccess != hipMalloc((void ***)&pdvert, num_polygons * sizeof(float*))) fprintf(stderr, "could not allocate array\n");

    if (hipSuccess != hipMalloc((void **)&cd_p1, num_contacts * sizeof(int))) fprintf(stderr, "could not allocate array\n");
    if (hipSuccess != hipMalloc((void **)&cd_p2, num_contacts * sizeof(int))) fprintf(stderr, "could not allocate array\n");
    if (hipSuccess != hipMalloc((void **)&cd_n_x, num_contacts * sizeof(float))) fprintf(stderr, "could not allocate array\n");
    if (hipSuccess != hipMalloc((void **)&cd_n_y, num_contacts * sizeof(float))) fprintf(stderr, "could not allocate array\n");
    if (hipSuccess != hipMalloc((void **)&cd_pen, num_contacts * sizeof(float))) fprintf(stderr, "could not allocate array\n");
    if (hipSuccess != hipMalloc((void **)&cd_used_flag, num_contacts * sizeof(int))) fprintf(stderr, "could not allocate array\n");

    if (hipSuccess != hipMemcpy(pd_x, polygon_x, num_polygons * sizeof(float), hipMemcpyHostToDevice)) fprintf(stderr, "copying of polygon_x to device failed\n");
    if (hipSuccess != hipMemcpy(pd_y, polygon_y, num_polygons * sizeof(float), hipMemcpyHostToDevice)) fprintf(stderr, "copying of polygon_y to device failed\n");
    if (hipSuccess != hipMemcpy(pd_num_vert, polygon_num_vertices, num_polygons * sizeof(int), hipMemcpyHostToDevice)) fprintf(stderr, "copying of polygon_num_vertices to device failed\n");
    if (hipSuccess != hipMemcpy(pdvert, polygon_vertices, num_polygons * sizeof(float*), hipMemcpyHostToDevice)) fprintf(stderr, "copying of polygon_vertices to device failed\n");
    detectCollisions<<<blocks, THREADS>>>(num_polygons, pd_x, pd_y, pd_num_vert, pdvert, cd_p1, cd_p2, cd_n_x, cd_n_y, cd_pen, cd_used_flag);
    if (hipSuccess != hipMemcpy(contact_p1, cd_p1, num_contacts * sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of cd_p1 from device failed\n");
    if (hipSuccess != hipMemcpy(contact_p2, cd_p2, num_contacts * sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of cd_p2 from device failed\n");
    if (hipSuccess != hipMemcpy(contact_n_x, cd_n_x, num_contacts * sizeof(float), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of cd_n_x from device failed\n");
    if (hipSuccess != hipMemcpy(contact_n_y, cd_n_y, num_contacts * sizeof(float), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of cd_n_y from device failed\n");
    if (hipSuccess != hipMemcpy(contact_penetration, cd_pen, num_contacts * sizeof(float), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of cd_pen from device failed\n");
    if (hipSuccess != hipMemcpy(contact_used_flag, cd_used_flag, num_contacts * sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of cd_used_flag from device failed\n");
    /* Execute pthread_init */
    pthread_init(polygon_x, polygon_y, contact_p1, contact_p2, contact_n_x, contact_n_y, contact_penetration, contact_used_flag, num_polygons, num_contacts, num_threads);

    /* End Time */
    gettimeofday(&end, NULL);

    /* Output Result */
    printf("runtime: %.4lf s\n", end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0);

    // free(polygons);
    // free(contacts);
    return 0;
}